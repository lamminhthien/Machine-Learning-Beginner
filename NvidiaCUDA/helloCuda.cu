
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <iostream>
#include<vector>
#include <cstdio>
using namespace std;
using namespace std::chrono;

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    for (int i=0; i<999;i++)
    {
        cuda_hello<<<1,256>>>();
    }
    return 0;
}
